#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include "hip/hip_runtime.h"
#include ""
#include "data_structures.h"
#include "utils.cuh"

// 统计各个表完成扫描的总数据量
__device__ int scanCustomerCounter = 0;
__device__ int scanOrderCounter = 0;
__device__ int scanLineitemCounter = 0;

// GPU 加速扫描 Customer 表
__global__ void scanCustomer(Customer *data, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size)
        return;

    int localCounter = 0;
    int localKeySum = 0;

    // Some computing task
    localKeySum += data[idx].C_CUSTKEY;
    localCounter += 1;

    atomicAdd(&scanCustomerCounter, localCounter);
}

// GPU 加速扫描 Orders 表
__global__ void scanOrders(Orders *data, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size)
        return;

    int localCounter = 0;
    int localPriceSum = 0;

    // Some computing task
    localPriceSum += (int)data[idx].O_TOTALPRICE;
    localCounter += 1;

    atomicAdd(&scanOrderCounter, localCounter);
}

// GPU 加速扫描 Lineitems 表
__global__ void scanLineitems(Lineitem *data, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= size)
        return;

    // Some computing task
    int localCounter = 0;
    int localQualitySum = 0;

    localQualitySum += data[idx].L_QUANTITY;
    localCounter += 1;

    atomicAdd(&scanLineitemCounter, localCounter);
}

template <typename T>
void checkScanResults(const T &deviceCounter, const char *tableName)
{
    int hostCounter;
    // 从设备内存中复制计数器值到主机内存
    checkCudaError(hipMemcpyFromSymbol(&hostCounter, HIP_SYMBOL(deviceCounter), sizeof(int), 0, hipMemcpyDeviceToHost));
    // 输出扫描得到的结果，包括表名
    std::cout << "Total scanned elements [Table-" << tableName << "]: " << hostCounter << std::endl;
}

int main()
{
    injectData();

    // 计算 BlockSize BlockNums
    dim3 blockSize(512);

    //每次扫描表
    dim3 numBlocksCustomers((customers.size() + blockSize.x - 1) / blockSize.x);
    dim3 numBlocksOrders((orders.size() + blockSize.x - 1) / blockSize.x);
    dim3 numBlocksLineitems((lineitems.size() + blockSize.x - 1) / blockSize.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float totalMilliseconds = 0;
    float milliseconds = 0;

    Customer *dev_customers = nullptr;
    Orders *dev_orders = nullptr;
    Lineitem *dev_lineitems = nullptr;

    // Record start time
    hipEventRecord(start);

    allocateAndLaunchKernel(
        (void *&)dev_customers,
        customers.data(),
        customers.size(),
        sizeof(Customer),
        blockSize,
        numBlocksCustomers,
        (void (*)(void *, int))scanCustomer);

    allocateAndLaunchKernel(
        (void *&)dev_orders,
        orders.data(),
        orders.size(),
        sizeof(Orders),
        blockSize,
        numBlocksOrders,
        (void (*)(void *, int))scanOrders);

    allocateAndLaunchKernel(
        (void *&)dev_lineitems,
        lineitems.data(),
        lineitems.size(),
        sizeof(Lineitem),
        blockSize,
        numBlocksLineitems,
        (void (*)(void *, int))scanLineitems);

    // Record stop time and calculate total time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    totalMilliseconds += milliseconds;

    checkScanResults(scanCustomerCounter, "Customer");
    checkScanResults(scanOrderCounter, "Order");
    checkScanResults(scanLineitemCounter, "LineItem");

    std::cout << "[Scan]Total time taken by GPU for all scans: " << totalMilliseconds << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}